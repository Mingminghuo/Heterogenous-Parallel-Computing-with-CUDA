#include<stdio.h>
#include<hip/hip_runtime.h>

#define NUM 327133

//Check Error
#define printError(func)                                                \
{                                                                       \
  hipError_t E  = func;                                                \
  if(E != hipSuccess)                                                  \
  {                                                                     \
    printf( "\nError at line: %d ", __LINE__);                          \
    printf( "\nError:  %s ", hipGetErrorString(E));                    \
  }                                                                     \
}                                                                       \

//Kernel
__global__ void add(float* A, float* B, float* C)
{
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<NUM)
  C[i] = B[i] + A[i];
}

//To check the output to see if it matches
int checkSum(float* A, float* B, float* C)
{
  for(int i = 0; i<NUM; i++)
    if(C[i] != A[i] + B[i])
      return 0;

  return 1;
}

int main()
{
  float* A;
  float* B;
  float* C;

  float* deviceA;
  float* deviceB;
  float* deviceC;

  A = (float*) malloc( NUM * sizeof(float));
  B = (float*) malloc( NUM * sizeof(float));
  C = (float*) malloc( NUM * sizeof(float));

  for(int i=0; i<NUM; i++)
  {
    A[i] = rand();
    B[i] = rand();
  }

  printError(hipMalloc((void **)&deviceA,  NUM * sizeof(float)));
  printError(hipMalloc((void **)&deviceB,  NUM * sizeof(float)));
  printError(hipMalloc((void **)&deviceC,  NUM * sizeof(float)));

  //cudaMalloc((void **)&deviceA,  NUM * sizeof(int));
  //cudaMalloc((void **)&deviceB,  NUM * sizeof(int));
  //cudaMalloc((void **)&deviceC,  NUM * sizeof(int));

  hipMemcpy(deviceA, A, NUM * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, B, NUM * sizeof(float), hipMemcpyHostToDevice);

  add<<<ceil(NUM/1024.0), 1024>>>(deviceA, deviceB, deviceC);

  hipMemcpy(C, deviceC, NUM * sizeof(float), hipMemcpyDeviceToHost);

  if(checkSum(A, B, C))
    printf("\nResult of 2 array sum is correct\n");

   else
     printf("\nResult of 2 array sum is wrong\n");

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  free(A);
  free(B);
  free(C);
}
