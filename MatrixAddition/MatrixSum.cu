#include<stdio.h>
#include<hip/hip_runtime.h>

#define ROW 100
#define COL 1000
//Check Error
#define printError(func)                                                \
{                                                                       \
  hipError_t E  = func;                                                \
  if(E != hipSuccess)                                                  \
  {                                                                     \
    printf( "\nError at line: %d ", __LINE__);                          \
    printf( "\nError:  %s ", hipGetErrorString(E));                    \
  }                                                                     \
}                                                                       \

//Kernel
__global__ void add(int A[][COL], int B[][COL], int C[][COL])
{
  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
  if( x < ROW && y < COL )
    C[x][y] = B[x][y] + A[x][y];
}

//To check the output to see if it matches
int checkSum(int A[][COL], int B[][COL], int C[][COL])
{
  for(int i = 0; i<ROW; i++)
    for(int j = 0; j<COL; j++)
      if(C[i][j] != A[i][j] + B[i][j])
        return 0;

  return 1;
}

int main()
{
  int A[ROW][COL];
  int B[ROW][COL];
  int C[ROW][COL];

  int (*deviceA)[COL];
  int (*deviceB)[COL];
  int (*deviceC)[COL];

  for(int i=0; i<ROW; i++)
  {
    for(int j=0; j<COL; j++)
    {
      A[i][j] = rand()%1000;
      B[i][j] = rand()%1000;
    }
  }

  printError(hipMalloc((void **)&deviceA,  ROW * COL * sizeof(int)));
  printError(hipMalloc((void **)&deviceB,  ROW * COL * sizeof(int)));
  printError(hipMalloc((void **)&deviceC,  ROW * COL * sizeof(int)));

  hipMemcpy(deviceA, A, ROW * COL * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, B, ROW * COL * sizeof(int), hipMemcpyHostToDevice);

  dim3 local_size(8, 8);
  dim3 global_size(ceil(ROW/8.0), ceil(COL/8.0));

  add<<<global_size, local_size>>>(deviceA, deviceB, deviceC);

  hipMemcpy(C, deviceC, ROW * COL * sizeof(int), hipMemcpyDeviceToHost);

/*  for(int i=0; i<ROW; i++)
  {
    for(int j=0; j<COL; j++)
    {
      printf("%d : %d, ", A[i][j] + B[i][j], C[i][j]);
    }
    printf("\n");
  }
*/
  if(checkSum(A, B, C))
    printf("\nResult of 2 matrix sum is correct\n");

   else
     printf("\nResult of 2 matrix sum is wrong\n");

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
}
